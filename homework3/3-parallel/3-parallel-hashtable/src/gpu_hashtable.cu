#include "hip/hip_runtime.h"
#include <iostream>
#include <limits.h>
#include <stdlib.h>
#include <string.h>
#include <stdio.h>
#include <ctime>
#include <sstream>
#include <string>
#include "test_map.hpp"
#include "gpu_hashtable.hpp"

using namespace std;

__host__ __device__ unsigned int hash_function_int(void *a)
{
	/*
	 * Credits: https://stackoverflow.com/a/12996028/7883884
	 * SD Laboratory
	 */
	unsigned int uint_a = *((unsigned int *)a);

	uint_a = ((uint_a >> 16u) ^ uint_a) * 0x45d9f3b;
	uint_a = ((uint_a >> 16u) ^ uint_a) * 0x45d9f3b;
	uint_a = (uint_a >> 16u) ^ uint_a;
	return uint_a;
}

/*
Allocate CUDA memory only through glbGpuAllocator
hipMalloc -> glbGpuAllocator->_cudaMalloc
hipMallocManaged -> glbGpuAllocator->_cudaMallocManaged
hipFree -> glbGpuAllocator->_cudaFree
*/

/**
 * Function constructor GpuHashTable
 * Performs init
 * Example on using wrapper allocators _cudaMalloc and _cudaFree
 */
GpuHashTable::GpuHashTable(int size) {
	this->size = 0;
	this->hmax = size;
	this->buckets = NULL;

	// Allocate memory (GPU/VRAM) for buckets
	glbGpuAllocator->_cudaMalloc((void **)&(this->buckets),
									size * sizeof(struct data));
	if (this->buckets == NULL) {
		DIE(1, "Could not allocate memory");
	}
}

/**
 * Function desctructor GpuHashTable
 */
GpuHashTable::~GpuHashTable() {
	glbGpuAllocator->_cudaFree(this->buckets);
}



__global__ void kernel_resize(struct data *old_buckets, struct data *new_buckets,
								int size, int old_hmax, int new_hmax) {

	int index = blockIdx.x * blockDim.x + threadIdx.x;
	if (index >= old_hmax) {
		return;
	}

	int key = old_buckets[index].key;
	if (key == 0) {
		return;
	}

	int val = old_buckets[index].value;
	int pos = hash_function_int(&key) % new_hmax;

	int compare_and_swap = atomicCAS(&(new_buckets[pos].key), 0, key);

	// Case 0 : Empty bucket => insert key:value (atomic operation)
	if (compare_and_swap == 0) {
		new_buckets[pos].value = val;
	} else {
		// Case 1 : Collision => find the next empty bucket
		while (atomicCAS(&(new_buckets[pos].key), 0, key) != 0) {
			pos = (pos + 1) % new_hmax;
		}
		new_buckets[pos].value = val;
	}
}


/**
 * Function reshape
 * Performs resize of the hashtable based on load factor
 */
void GpuHashTable::reshape(int numBucketsReshape) {
	struct data *new_buckets = NULL;
	glbGpuAllocator->_cudaMalloc((void **)&(new_buckets),
									numBucketsReshape * sizeof(struct data));
	int new_hmax = numBucketsReshape;

	// Parallelize the copy of the old buckets to the new ones
	int blocks = this->hmax / 256;
	int threads = 256;
	if (this->hmax % 256 != 0) {
		blocks++;
	}
	kernel_resize<<<blocks, threads>>>(this->buckets, new_buckets,
									this->size, this->hmax, new_hmax);
	hipDeviceSynchronize();

	// Update the fields of the hashtable
	struct data *old_buckets = this->buckets;
	this->buckets = new_buckets;
	this->hmax = new_hmax;

	glbGpuAllocator->_cudaFree(old_buckets);

	return;
}


__global__ void kernel_insert(int *keys, int *value, int numKeys,
								struct data *buckets, int size, int hmax) {

	int index = blockIdx.x * blockDim.x + threadIdx.x;
	if (index >= numKeys) {
		return;
	}

	// (key, value) -> data to be inserted
	int key = keys[index];
	int val = value[index];
	int pos = hash_function_int(&key) % hmax;
	int curr_pos = 0, ref_pos = 0;
	
	// Atomic operation to see if the bucket is empty
	int compare_and_swap = atomicCAS(&(buckets[pos].key), 0, key);

	// Case 0 : Empty bucket => insert key:value (atomic operation)
	// Case 1 : Key already exists => update value
	if (compare_and_swap == 0 || compare_and_swap == key) {
		//atomicExch(&buckets[pos].value, val);
		buckets[pos].value = val;
		return;

	}

	// Case2: Collision
	ref_pos = pos;
	curr_pos = (pos + 1) % hmax;
	while (curr_pos != ref_pos) {
		compare_and_swap = atomicCAS(&(buckets[curr_pos].key), 0, key);

		// Case 2.1: key already exists but in another bucket -> update value
		// Case 2.2: key doesn't exist -> old key is 0
		if (compare_and_swap == key || compare_and_swap == 0) {
			//atomicExch(&buckets[curr_pos].value, val);
			buckets[curr_pos].value = val;
			return;
		}
		curr_pos = (curr_pos + 1) % hmax;
	}
}

/**
 * Function insertBatch
 * Inserts a batch of key:value, using GPU and wrapper allocators
 */
bool GpuHashTable::insertBatch(int *keys, int* values, int numKeys) {
	int new_size = 0;

	// In case of not enough space, resize the hashtable
	float old_factor = (float)(this->size + numKeys) / (float)this->hmax;
	if (old_factor > this->max_threshold) {
		new_size = (this->size + numKeys) /this->regular_threshold;
		this->reshape(new_size);
	}

	// Allocate memory on GPU for keys and values
	int *d_keys = NULL;
	int *d_values = NULL;
	glbGpuAllocator->_cudaMalloc((void **)&(d_keys), numKeys * sizeof(int));
	hipMemcpy(d_keys, keys, numKeys * sizeof(int), hipMemcpyHostToDevice);

	glbGpuAllocator->_cudaMalloc((void **)&(d_values), numKeys * sizeof(int));
	hipMemcpy(d_values, values, numKeys * sizeof(int), hipMemcpyHostToDevice);


	// Insert the batch of keys and values
	int blocks = numKeys / 256;
	int threads = 256;
	if (numKeys % 256 != 0) {
		blocks++;
	}
	kernel_insert<<<blocks, threads>>>(d_keys, d_values, numKeys, this->buckets,
										this->size, this->hmax);
	hipDeviceSynchronize();

	this->size += numKeys;

	// Free memory on GPU
	glbGpuAllocator->_cudaFree(d_keys);
	glbGpuAllocator->_cudaFree(d_values);

	return true;
}


__global__ void kernel_get_batch(int *keys, int num, struct data *buckets,
										int size, int hmax, int *result_vec) {
    
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	if (index >= num) {
		return;
	}


	int key = keys[index];
	int pos = hash_function_int(&key) % hmax;
	int result = -1;

	if (buckets[pos].key == key) {
		result = buckets[pos].value;
		result_vec[index] = result;
		return;
	}

	int ref_pos = pos;
	int curr_pos = (pos + 1) % hmax;
	while (curr_pos != ref_pos) {
		if (buckets[curr_pos].key == key) {
			result = buckets[curr_pos].value;
			result_vec[index] = result;
			return;
		}
		// Can't find the key
		if (buckets[curr_pos].key == 0) {
			result_vec[index] = result;
			return;
		}
		curr_pos = (curr_pos + 1) % hmax;
	}
	result_vec[index] = result;
		
	return;
}


/**
 * Function getBatch
 * Gets a batch of key:value, using GPU
 */
int* GpuHashTable::getBatch(int* keys, int numKeys) {

	int blocks = numKeys / 256;
	int threads = 256;
	if (numKeys % 256 != 0) {
		blocks++;
	}

	// Allocate memory (GPU/VRAM) for result vector
	int *result_vec_gpu = NULL;
	glbGpuAllocator->_cudaMalloc((void **)&(result_vec_gpu), numKeys * sizeof(int));

	// Allocate memory (GPU/VRAM) for keys
	int *d_keys = NULL;
	glbGpuAllocator->_cudaMalloc((void **)&(d_keys), numKeys * sizeof(int));
	hipMemcpy(d_keys, keys, numKeys * sizeof(int), hipMemcpyHostToDevice);

	// Each CPU kernel will write the result in the result vector
	kernel_get_batch<<<blocks, threads>>>(d_keys, numKeys, this->buckets,
										this->size, this->hmax, result_vec_gpu);

	hipDeviceSynchronize();

	// The returned result vector will be copied from GPU memory to host memory
	int *result_vec_cpu = (int *)malloc(numKeys * sizeof(int));
	hipMemcpy(result_vec_cpu, result_vec_gpu,
				numKeys * sizeof(int),hipMemcpyDeviceToHost);
	
	// Free memory (GPU/VRAM) for result vector
	glbGpuAllocator->_cudaFree(result_vec_gpu);
	glbGpuAllocator->_cudaFree(d_keys);

	// Final result from RAM
	return result_vec_cpu;
}