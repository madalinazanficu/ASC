#include "hip/hip_runtime.h"
#include <iostream>
#include <limits.h>
#include <stdlib.h>
#include <string.h>
#include <stdio.h>
#include <ctime>
#include <sstream>
#include <string>
#include "test_map.hpp"
#include "gpu_hashtable.hpp"

using namespace std;

unsigned int hash_function_int(void *a)
{
	/*
	 * Credits: https://stackoverflow.com/a/12996028/7883884
	 */
	unsigned int uint_a = *((unsigned int *)a);

	uint_a = ((uint_a >> 16u) ^ uint_a) * 0x45d9f3b;
	uint_a = ((uint_a >> 16u) ^ uint_a) * 0x45d9f3b;
	uint_a = (uint_a >> 16u) ^ uint_a;
	return uint_a;
}

/*
Allocate CUDA memory only through glbGpuAllocator
hipMalloc -> glbGpuAllocator->_cudaMalloc
hipMallocManaged -> glbGpuAllocator->_cudaMallocManaged
hipFree -> glbGpuAllocator->_cudaFree
*/

/**
 * Function constructor GpuHashTable
 * Performs init
 * Example on using wrapper allocators _cudaMalloc and _cudaFree
 */
GpuHashTable::GpuHashTable(int size) {
	this->size = 0;
	this->hmax = size;
	this->buckets = NULL;

	// Allocate memory (GPU/VRAM) for buckets
	glbGpuAllocator->_cudaMalloc((void *)&this->buckets, size * sizeof(struct data));
	if (this->buckets == NULL) {
		printf("Could not allocate memory");
	}
}

/**
 * Function desctructor GpuHashTable
 */
GpuHashTable::~GpuHashTable() {
	glbGpuAllocator->_cudaFree(this->buckets);
}


/**
 * Function reshape
 * Performs resize of the hashtable based on load factor
 */
void GpuHashTable::reshape(int numBucketsReshape) {

	// Allocate new memory (GPU/VRAM) for more buckets
	struct data *new_buckets = NULL;
	glbGpuAllocator->_cudaMalloc((void *)&new_buckets,
									numBucketsReshape * sizeof(struct data));

	// Copy data from old buckets to new buckets
	// glbGpuAllocator->_cudaMemcpy(new_buckets, this->buckets,
	// 								this->size * sizeof(struct data),
	// 								hipMemcpyDeviceToDevice);

	hipMemcpy(new_buckets, this->buckets,
				this->size * sizeof(struct data),
				hipMemcpyDeviceToDevice);

	// Free old buckets
	glbGpuAllocator->_cudaFree(this->buckets);

	// Update hashtable fields
	this->buckets = new_buckets;
	this->hmax = numBucketsReshape;

	return;
}


__global__ void kernel_insert(int *keys, int *value, int numKeys,
								struct data *buckets, int size, int hmax) {

	int index = blockIdx.x * blockDim.x + threadIdx.x;

	int key = keys[index];
	int val = value[index];
	int pos = hash_function_int(&key) % hmax;

	switch buckets[pos].key {
	
	// Empty bucket => insert key:value
	case 0:
		buckets[pos].key = key;
		buckets[pos].value = val;
		break;

	// Key already exists => update value
	case key:
		buckets[pos].value = val;
		break;

	// Collision
	default:
		
		// Case 3.0: key already exists but in another bucket => update value
		int ref_pos = pos;
		int curr_pos = (pos + 1) % hmmax;
		bool stop = false;
		while (curr_pos != ref_pos) {
			if (buckets[curr_pos].key == key) {
				buckets[curr_pos].value = val;
				stop = true;
				break;
			}
			curr_pos = (curr_pos + 1) % size;
		}
		if (stop == true) {
			break;
		}

		// Case 3.1: find the next available slot
		while (buckets[pos].key != 0) {
			pos = (pos + 1) % size;
		}
		buckets[pos].key = key;
		buckets[pos].value = val;
		break;
	}
}

/**
 * Function insertBatch
 * Inserts a batch of key:value, using GPU and wrapper allocators
 */
bool GpuHashTable::insertBatch(int *keys, int* values, int numKeys) {

	int available_space = this->hmax - this->size;

	// In case of not enough space, resize the hashtable
	if (available_space <= numKeys) {
		int new_size = this->hmax + numKeys;
		this->reshape(new_size);
	}

	int blocks = numKeys / 256;
	int threads = 256;
	kernel_insert<<<blocks, threads>>>(keys, values, numKeys, this->buckets,
										this->size, this->hmax);

	return true;
}

__global__ void kernel_get_batch(int *keys, int num, struct data *buckets,
										int size, int hmax, int *result_vec) {
    
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	int key = keys[index];
	int pos = hash_function_int(&key) % hmax;
	int result = 0;

	switch buckets[pos].key {
	
	// The key is in the right bucket
	case key:
		result = buckets[pos].value;
		break;

	// A collision occured, so we need to search for the key
	default:
		int ref_pos = pos;
		int curr_pos = (pos + 1) % hmmax;
		while (curr_pos != ref_pos) {
			if (buckets[curr_pos].key == key) {
				result = buckets[curr_pos].value;
				break;
			}
			curr_pos = (curr_pos + 1) % size;
		}
		break;
	}
	result_vec[index] = result;
}

/**
 * Function getBatch
 * Gets a batch of key:value, using GPU
 */
int* GpuHashTable::getBatch(int* keys, int numKeys) {

	int blocks = numKeys / 256;
	int threads = 256;

	// Allocate memory (GPU/VRAM) for result vector
	int *result_vec_gpu = NULL;
	glbGpuAllocator->_cudaMalloc((void *)&result_vec, numKeys * sizeof(int));


	// Each CPU kernel will write the result in the result vector
	kernel_get_batch<<<blocks, threads>>>(keys, numKeys, this->buckets,
										this->size, this->hmax, result_vec_gpu);


	// The returned result vector will be copied from GPU memory to host memory
	int result_vec_cpu = malloc(numKeys * sizeof(int));
	// glbGpuAllocator->_cudaMemcpy(result_vec_cpu, result_vec_gpu,
	// 								numKeys * sizeof(int),hipMemcpyDeviceToHost);

	hipMemcpy(result_vec_cpu, result_vec_gpu,
				numKeys * sizeof(int),hipMemcpyDeviceToHost);
	
	// Free memory (GPU/VRAM) for result vector
	glbGpuAllocator->_cudaFree(result_vec_gpu);

	return result_vec_cpu;
}
