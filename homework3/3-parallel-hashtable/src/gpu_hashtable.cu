#include "hip/hip_runtime.h"
#include <iostream>
#include <limits.h>
#include <stdlib.h>
#include <string.h>
#include <stdio.h>
#include <ctime>
#include <sstream>
#include <string>
#include "test_map.hpp"
#include "gpu_hashtable.hpp"

using namespace std;

__host__ __device__ unsigned int hash_function_int(void *a)
{
	/*
	 * Credits: https://stackoverflow.com/a/12996028/7883884
	 */
	unsigned int uint_a = *((unsigned int *)a);

	uint_a = ((uint_a >> 16u) ^ uint_a) * 0x45d9f3b;
	uint_a = ((uint_a >> 16u) ^ uint_a) * 0x45d9f3b;
	uint_a = (uint_a >> 16u) ^ uint_a;
	return uint_a;
}

/*
Allocate CUDA memory only through glbGpuAllocator
hipMalloc -> glbGpuAllocator->_cudaMalloc
hipMallocManaged -> glbGpuAllocator->_cudaMallocManaged
hipFree -> glbGpuAllocator->_cudaFree
*/

/**
 * Function constructor GpuHashTable
 * Performs init
 * Example on using wrapper allocators _cudaMalloc and _cudaFree
 */
GpuHashTable::GpuHashTable(int size) {
	this->size = 0;
	this->hmax = size;
	this->buckets = NULL;

	// Allocate memory (GPU/VRAM) for buckets

	//cout << "In constructor" << endl;

	glbGpuAllocator->_cudaMalloc((void **)&(this->buckets), size * sizeof(struct data));
	if (this->buckets == NULL) {
		printf("Could not allocate memory");
	}

	//cout << "End of constructor" << endl;
}

/**
 * Function desctructor GpuHashTable
 */
GpuHashTable::~GpuHashTable() {
	glbGpuAllocator->_cudaFree(this->buckets);
}


/**
 * Function reshape
 * Performs resize of the hashtable based on load factor
 */
void GpuHashTable::reshape(int numBucketsReshape) {

	cout << "In reshape" << endl;

	// Allocate new memory (GPU/VRAM) for more buckets
	struct data *new_buckets = NULL;
	glbGpuAllocator->_cudaMalloc((void **)&(new_buckets),
									numBucketsReshape * sizeof(struct data));
	
	// Save the reference of the old buckets
	struct data *old_buckets = this->buckets;

	// Get all the keys and values from the old buckets
	int *keys = getAllKeys(this->size);
	int *values = getBatch(keys, this->size);
	int num_keys = this->size;

	// Update the hashtable fields
	this->hmax = numBucketsReshape;
	this->buckets = new_buckets;
	this->size = 0;

	// Insert all the elements from the old buckets to the new one
	insertBatch(keys, values, num_keys);

	// Free old buckets
	glbGpuAllocator->_cudaFree(old_buckets);


	cout << "IN reshape END" << endl << endl;
	return;
}


__global__ void kernel_insert(int *keys, int *value, int numKeys,
								struct data *buckets, int size, int hmax) {

	int index = blockIdx.x * blockDim.x + threadIdx.x;
	if (index >= numKeys) {
		return;
	}

	// (key, value) -> data to be inserted
	int key = keys[index];
	int val = value[index];
	int pos = hash_function_int(&key) % hmax;
	int curr_pos = 0, ref_pos = 0;
	bool stop = false;

	
	int compare_and_swap = atomicCAS(&(buckets[pos].key), 0, key);

	// Case 0 : Empty bucket => insert key:value (atomic operation)
	// Case 1 : Key already exists => update value
	if (compare_and_swap == 0 || compare_and_swap == key) {
		buckets[pos].value = val;
		return;

	} else {
		// Case 2: key already exists but in another bucket
		ref_pos = pos;
		curr_pos = (pos + 1) % hmax;
		stop = false;
		while (curr_pos != ref_pos) {
			if (buckets[curr_pos].key == key) {
				buckets[curr_pos].value = val;
				stop = true;
				break;
			}
			curr_pos = (curr_pos + 1) % size;
		}
		if (stop == true) {
			return;
		}

		// Case 3: find the next available slot (atomic operation)
		while (atomicCAS(&(buckets[pos].key), 0, key) != 0) {
			pos = (pos + 1) % hmax;
		}
		buckets[pos].value = val;
	}
}

/**
 * Function insertBatch
 * Inserts a batch of key:value, using GPU and wrapper allocators
 */
bool GpuHashTable::insertBatch(int *keys, int* values, int numKeys) {

	//cout << "In insertBatch" << endl;

	//int available_space = this->hmax - this->size;
	// if (available_space <= numKeys) {
	// 	int new_size = (this->hmax + numKeys) * 3;
	// 	this->reshape(new_size);
	// }
	int new_size = 0;
	double new_factor = 0.0;

	// In case of not enough space, resize the hashtable
	double old_factor = (this->size + numKeys) / this->hmax;
	cout << "Old hmax: " << this->hmax << endl;
	cout << "Num keys: " << numKeys << endl;
	cout << "Old size: " << this->size << endl;
	cout << "Old factor: " << old_factor << endl;
	if (old_factor > 0.8) {
		new_factor = 0.5;
		new_size = (this->size + numKeys) / new_factor;
		this->reshape(new_size);
	}

	// Allocate memory on GPU for keys and values
	int *d_keys = NULL;
	int *d_values = NULL;
	glbGpuAllocator->_cudaMalloc((void **)&(d_keys), numKeys * sizeof(int));
	hipMemcpy(d_keys, keys, numKeys * sizeof(int), hipMemcpyHostToDevice);

	glbGpuAllocator->_cudaMalloc((void **)&(d_values), numKeys * sizeof(int));
	hipMemcpy(d_values, values, numKeys * sizeof(int), hipMemcpyHostToDevice);


	// Insert the batch of keys and values
	int blocks = numKeys / 256;
	int threads = 256;
	kernel_insert<<<blocks, threads>>>(d_keys, d_values, numKeys, this->buckets,
										this->size, this->hmax);
	hipDeviceSynchronize();

	this->size += numKeys;

	cout << "New hmax: " << this->hmax << endl;
	cout << "Num keys: " << numKeys << endl;
	cout << "New size: " << this->size << endl;
	cout << "New factor: " << new_factor << endl;

	//cout << "End of insertBatch" << endl;
	return true;
}

__global__ void kernel_get_batch(int *keys, int num, struct data *buckets,
										int size, int hmax, int *result_vec) {
    
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	if (index >= num) {
		return;
	}


	int key = keys[index];
	int pos = hash_function_int(&key) % hmax;
	int result = 0;

	if (buckets[pos].key == key) {
		result = buckets[pos].value;
		result_vec[index] = result;
		return;
	}

	int ref_pos = pos;
	int curr_pos = (pos + 1) % hmax;
	while (curr_pos != ref_pos) {
		if (buckets[curr_pos].key == key) {
			result = buckets[curr_pos].value;
			break;
		}
		curr_pos = (curr_pos + 1) % size;
	}
	result_vec[index] = result;
		
	return;
}

/**
 * Function getBatch
 * Gets a batch of key:value, using GPU
 */
int* GpuHashTable::getBatch(int* keys, int numKeys) {

	//cout << "In getBatch" << endl;

	int blocks = numKeys / 256;
	int threads = 256;

	// Allocate memory (GPU/VRAM) for result vector
	int *result_vec_gpu = NULL;
	glbGpuAllocator->_cudaMalloc((void **)&(result_vec_gpu), numKeys * sizeof(int));


	// Each CPU kernel will write the result in the result vector
	kernel_get_batch<<<blocks, threads>>>(keys, numKeys, this->buckets,
										this->size, this->hmax, result_vec_gpu);

	hipDeviceSynchronize();

	// The returned result vector will be copied from GPU memory to host memory
	int *result_vec_cpu = (int *)malloc(numKeys * sizeof(int));
	hipMemcpy(result_vec_cpu, result_vec_gpu,
				numKeys * sizeof(int),hipMemcpyDeviceToHost);
	
	// Free memory (GPU/VRAM) for result vector
	glbGpuAllocator->_cudaFree(result_vec_gpu);

	//cout << "In getBatch - END" << endl;

	// Final result from RAM
	return result_vec_cpu;
}



__global__ void kernel_get_keys(int numKeys, struct data *buckets,
									int size, int hmax, int *result_vec) {
	
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	if (index >= numKeys) {
		return;
	}

	int key = buckets[index].key;

	if (key != 0) {
		result_vec[index] = key;
	}
		
	return;
}

int* GpuHashTable::getAllKeys(int numKeys) {

	//cout << "In getAllKeys" << endl;

	int blocks = numKeys / 256;
	int threads = 256;

	// Allocate memory (GPU/VRAM) for result vector
	int *result_vec_gpu = NULL;
	glbGpuAllocator->_cudaMalloc((void **)&(result_vec_gpu), numKeys * sizeof(int));

	// Each CPU kernel will write the result in the result vector
	kernel_get_keys<<<blocks, threads>>>(numKeys, this->buckets,
										this->size, this->hmax, result_vec_gpu);
	hipDeviceSynchronize();

	int *result_vec_cpu = (int *)malloc(numKeys * sizeof(int));
	hipMemcpy(result_vec_cpu, result_vec_gpu,
				numKeys * sizeof(int),hipMemcpyDeviceToHost);

	// Free memory (GPU/VRAM) for result vector
	glbGpuAllocator->_cudaFree(result_vec_gpu);

	//cout << "In getAllKeys - END" << endl;

	// Final result from RAM
	return result_vec_cpu;
}
